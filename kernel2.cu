#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "Chi8.h"

__global__ void kernel( unsigned int rows, unsigned int cols , int cRows , int contRows ,int jobs,int ref, unsigned char *snpdata,float *results,int *dev_colid){
        unsigned char x, y;
        int m, n ;
	unsigned int p = 0 ;
        float Xmean = 0, Ymean = 0;
        float numerator=0;
	float Xvar = 0, Yvar = 0;

        int tid  = threadIdx.x + blockIdx.x * blockDim.x;
	
	if ((tid < jobs) && (tid > LD_width )) {

                for ( m = 0 ; m < cRows ; m++ ) {
                	x = snpdata[m * cols + ref];
                        y = snpdata[m * cols + (ref + tid)];

                        if ( x == '0' && y == '0') { }
                        else if (x == '0' && y == '1') { Xmean++; }
                        else if (x == '0' && y == '2') { Xmean+=2; }
                        else if (x == '1' && y == '0') { Xmean+=3; }
                        else if (x == '1' && y == '1') { Xmean+=4; }
                        else if (x == '1' && y == '2' ) { Xmean+=5; }
                        else if (x == '2' && y == '0' ) { Xmean+=6; }
                        else if (x == '2' && y == '1') { Xmean+=7; }
                        else if (x == '2' && y == '2') { Xmean+=8; }

			Ymean += 0;
                 }
               
		 for ( n = cRows ; n < cRows + contRows ; n++ ) {
                 	x = snpdata[n * cols + ref];
                        y = snpdata[n * cols + (ref + tid)];

                        if ( x == '0' && y == '0' ) { }
                        else if (x == '0' && y == '1') { Xmean++; }
                        else if (x == '0' && y == '2') { Xmean+=2; }
                        else if (x == '1' && y == '0') { Xmean+=3; }
                        else if (x == '1' && y == '1') { Xmean+=4; }
                        else if (x == '1' && y == '2' ) { Xmean+=5; }
                        else if (x == '2' && y == '0' ) { Xmean+=6; }
                        else if (x == '2' && y == '1') { Xmean+=7; }
                        else if (x == '2' && y == '2') { Xmean+=8; }

                        Ymean += 1;
                  }
		
		Xmean /= (cRows+contRows); Ymean /= (cRows+contRows);				
		
                for ( m = 0 ; m < cRows ; m++ ) {

                	x = snpdata[m * cols + ref];
                        y = snpdata[m * cols + (ref + tid)];

                        if ( x == '0' && y == '0') { numerator += (0 - Xmean)*(0 - Ymean); Xvar += (0 - Xmean)*(0 - Xmean); }
                        else if (x == '0' && y == '1') { numerator += (1 - Xmean)*(0 - Ymean); Xvar += (1 - Xmean)*(1 - Xmean); }
                        else if (x == '0' && y == '2') { numerator += (2 - Xmean)*(0 - Ymean); Xvar += (2 - Xmean)*(2 - Xmean); }
                        else if (x == '1' && y == '0') { numerator += (3 - Xmean)*(0 - Ymean); Xvar += (3 - Xmean)*(3 - Xmean); }
                        else if (x == '1' && y == '1') { numerator += (4 - Xmean)*(0 - Ymean); Xvar += (4 - Xmean)*(4 - Xmean); }
                        else if (x == '1' && y == '2' ) { numerator += (5 - Xmean)*(0 - Ymean); Xvar += (5 - Xmean)*(5 - Xmean); }
                        else if (x == '2' && y == '0' ) { numerator += (6 - Xmean)*(0 - Ymean); Xvar += (6 - Xmean)*(6 - Xmean); }
                        else if (x == '2' && y == '1') { numerator += (7 - Xmean)*(0 - Ymean); Xvar += (7 - Xmean)*(7 - Xmean); }
                        else if (x == '2' && y == '2') { numerator += (8 - Xmean)*(0 - Ymean); Xvar += (8 - Xmean)*(8 - Xmean); }
	
			Yvar += (0 - Ymean)*(0 - Ymean);
                 }
               
		 for ( n = cRows ; n < cRows + contRows ; n++ ) {
                 	x = snpdata[n * cols + ref];
                        y = snpdata[n * cols + (ref + tid)];

                        if ( x == '0' && y == '0') { numerator += (0 - Xmean)*(1 - Ymean); Xvar += (0 - Xmean)*(0 - Xmean); }
                        else if (x == '0' && y == '1') { numerator += (1 - Xmean)*(1 - Ymean); Xvar += (1 - Xmean)*(1 - Xmean); }
                        else if (x == '0' && y == '2') { numerator += (2 - Xmean)*(1 - Ymean); Xvar += (2 - Xmean)*(2 - Xmean); }
                        else if (x == '1' && y == '0') { numerator += (3 - Xmean)*(1 - Ymean); Xvar += (3 - Xmean)*(3 - Xmean); }
                        else if (x == '1' && y == '1') { numerator += (4 - Xmean)*(1 - Ymean); Xvar += (4 - Xmean)*(4 - Xmean); }
                        else if (x == '1' && y == '2' ) { numerator += (5 - Xmean)*(1 - Ymean); Xvar += (5 - Xmean)*(5 - Xmean); }
                        else if (x == '2' && y == '0' ) { numerator += (6 - Xmean)*(1 - Ymean); Xvar += (6 - Xmean)*(6 - Xmean); }
                        else if (x == '2' && y == '1') { numerator += (7 - Xmean)*(1 - Ymean); Xvar += (7 - Xmean)*(7 - Xmean); }
                        else if (x == '2' && y == '2') { numerator += (8 - Xmean)*(1 - Ymean); Xvar += (8 - Xmean)*(8 - Xmean); }

			Yvar += (1 - Ymean)*(1 - Ymean);
                  }
		Xvar = sqrt(Xvar); Yvar = sqrt(Yvar);
		dev_colid[tid] = tid;
 		results[tid] = abs(numerator/(Xvar*Yvar));
				
	}
}


#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "dot.h"


__global__ void kernel(unsigned int mtrows, unsigned int mtcols , float *mtdata, float *wdata,float *results){
/* exception if #Mrows!= #vectorpts */
	
	unsigned int x,tid;
	float sum = 0.0f;
	float results[mtrows];
	float temp[mtcols];


	for(tid = 0; tid < mtrows; tid++) {

		for(x=0;x<mtcols;x++){
			/*Do we need to specify the memory location using size_t*/
			temp[x] = wdata[x]* mtdata[tid][x];
			sum += temp[x];
		}

		results[tid]=sum;
	}
}